
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

// Parts of this code sourced from SnopyDogy
// https://gist.github.com/SnopyDogy/a9a22497a893ec86aa3e

#if defined(WITH_GRAPHICS)

#include <Array.hpp>
#include <image.hpp>
#include <err_cuda.hpp>
#include <debug_cuda.hpp>
#include <interopManager.hpp>

using af::dim4;

namespace cuda
{

template<typename T>
void copy_image(const Array<T> &in, const fg::Image* image)
{
    if(InteropManager::checkGraphicsInteropCapability()) {
        InteropManager& intrpMngr = InteropManager::getInstance();

        cudaGraphicsResource *cudaPBOResource = intrpMngr.getBufferResource(image);

        const T *d_X = in.get();
        // Map resource. Copy data to PBO. Unmap resource.
        size_t num_bytes;
        T* d_pbo = NULL;
        cudaGraphicsMapResources(1, &cudaPBOResource, 0);
        cudaGraphicsResourceGetMappedPointer((void **)&d_pbo, &num_bytes, cudaPBOResource);
        cudaMemcpyAsync(d_pbo, d_X, num_bytes, cudaMemcpyDeviceToDevice,
                        cuda::getStream(cuda::getActiveDeviceId()));
        cudaGraphicsUnmapResources(1, &cudaPBOResource, 0);

        POST_LAUNCH_CHECK();
        CheckGL("After cuda resource copy");
    } else {
        CheckGL("Begin CUDA fallback-resource copy");
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, image->pbo());
        glBufferData(GL_PIXEL_UNPACK_BUFFER, image->size(), 0, GL_STREAM_DRAW);
        GLubyte* ptr = (GLubyte*)glMapBuffer(GL_PIXEL_UNPACK_BUFFER, GL_WRITE_ONLY);
        if (ptr) {
            CUDA_CHECK(cudaMemcpy(ptr, in.get(), image->size(), cudaMemcpyDeviceToHost));
            glUnmapBuffer(GL_PIXEL_UNPACK_BUFFER);
        }
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);
        CheckGL("End CUDA fallback-resource copy");
    }
}

#define INSTANTIATE(T)      \
    template void copy_image<T>(const Array<T> &in, const fg::Image* image);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(uchar)
INSTANTIATE(char)
INSTANTIATE(ushort)
INSTANTIATE(short)

}

#endif
