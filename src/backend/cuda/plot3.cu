
#include <hip/hip_runtime.h>
/*******************************************************
 * Copyright (c) 2014, ArrayFire
 * All rights reserved.
 *
 * This file is distributed under 3-clause BSD license.
 * The complete license agreement can be obtained at:
 * http://arrayfire.com/licenses/BSD-3-Clause
 ********************************************************/

#if defined (WITH_GRAPHICS)

#include <interopManager.hpp>
#include <Array.hpp>
#include <plot3.hpp>
#include <err_cuda.hpp>
#include <debug_cuda.hpp>
#include <join.hpp>
#include <reduce.hpp>
#include <reorder.hpp>

using af::dim4;

namespace cuda
{

template<typename T>
void copy_plot3(const Array<T> &P, fg::Plot3* plot3)
{
    if(InteropManager::checkGraphicsInteropCapability()) {
        const T *d_P = P.get();

        InteropManager& intrpMngr = InteropManager::getInstance();

        cudaGraphicsResource *cudaVBOResource = intrpMngr.getBufferResource(plot3);
        // Map resource. Copy data to VBO. Unmap resource.
        size_t num_bytes = plot3->size();
        T* d_vbo = NULL;
        cudaGraphicsMapResources(1, &cudaVBOResource, 0);
        cudaGraphicsResourceGetMappedPointer((void **)&d_vbo, &num_bytes, cudaVBOResource);
        cudaMemcpyAsync(d_vbo, d_P, num_bytes, cudaMemcpyDeviceToDevice,
                cuda::getStream(cuda::getActiveDeviceId()));
        cudaGraphicsUnmapResources(1, &cudaVBOResource, 0);

        CheckGL("After cuda resource copy");

        POST_LAUNCH_CHECK();
    } else {
        CheckGL("Begin CUDA fallback-resource copy");
        glBindBuffer(GL_ARRAY_BUFFER, plot3->vbo());
        GLubyte* ptr = (GLubyte*)glMapBuffer(GL_ARRAY_BUFFER, GL_WRITE_ONLY);
        if (ptr) {
            CUDA_CHECK(cudaMemcpy(ptr, P.get(), plot3->size(), cudaMemcpyDeviceToHost));
            glUnmapBuffer(GL_ARRAY_BUFFER);
        }
        glBindBuffer(GL_ARRAY_BUFFER, 0);
        CheckGL("End CUDA fallback-resource copy");
    }
}

#define INSTANTIATE(T)  \
    template void copy_plot3<T>(const Array<T> &P, fg::Plot3* plot3);

INSTANTIATE(float)
INSTANTIATE(double)
INSTANTIATE(int)
INSTANTIATE(uint)
INSTANTIATE(short)
INSTANTIATE(ushort)
INSTANTIATE(uchar)

}

#endif  // WITH_GRAPHICS
